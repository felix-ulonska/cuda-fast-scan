#include "hip/hip_runtime.h"
#include <iostream>
#include "params.cuh"
#include "main.cuh"
#include <cstdio>

// CURSED ONE-File
void scan_host(int *dest, int *src, int n) {
  int *currDest = dest;
  int *currSrc = src;

  // Setting first value
  *currDest = *currSrc;

  // Moving the pointers through the array and using the last value to calc the
  // next value
  do {
    int nextVal = *(++currSrc) + *currDest;
    *(++currDest) = nextVal;
  } while (currDest != &dest[n]);
}

bool arr_equal(int *a, int *b, int n) {
  bool bad = false;
  for (int i = 0; i < n; i++) {
    if (abs(a[i] - b[i]) > 0.3) {
      std::cout << "I: " << i << " First val: " << a[i] << " Second Val" << b[i]
                << std::endl;
      return false;
      bad = true;
    }
  }
  return !bad;
}

__global__ void scan_kernel(int* input, int* flag, int* agg, int* prefix) {
  extern __shared__ int s[];

  // Parfor block
  {
    int* b_ptr_input = &input[ITEMS_PER_BLOCK * blockIdx.x];
    int* b_ptr_shared_reduction = &s[0];
    int* b_ptr_shared_input_copy = &s[blockDim.x];

    for (int i = 0; i < ITEMS_PER_BLOCK; i += blockDim.x) {
      b_ptr_shared_input_copy[i + threadIdx.x] = b_ptr_input[i + threadIdx.x];
    }
    __syncthreads();


    // Parfor thread in block
    int* t_ptr_input = &b_ptr_input[threadIdx.x * ITEMS_PER_THREAD];
    int* t_ptr_shared_reduction = &b_ptr_shared_reduction[threadIdx.x];
    int* t_ptr_shared_input = &b_ptr_shared_input_copy[threadIdx.x * ITEMS_PER_THREAD];
    // PARFOR THREAD
    {
      // t_mem_cpy(t_ptr_shared_input, t_ptr_input);
			int sum = t_ptr_shared_input[0];
			for (int i = 1; i < ITEMS_PER_THREAD; i++) {
				sum = sum+ t_ptr_shared_input[i];
			} 
			*t_ptr_shared_reduction = sum;
		}
		__syncthreads();

		for (std::size_t k = (THREADS_PER_BLOCK / 2); k > 0; k = k / 2) {
			if (threadIdx.x < k) {
				b_ptr_shared_reduction[(threadIdx.x - 0)] =
						b_ptr_shared_reduction[(threadIdx.x - 0)] +
						b_ptr_shared_reduction[((threadIdx.x - 0) + k)];
			}
			__syncthreads();
		}

		if (threadIdx.x == 0) {
			if (blockIdx.x != 0) {
				agg[blockIdx.x] = b_ptr_shared_reduction[0];       
				__threadfence();
				flag[blockIdx.x] = 1;
			} else if (threadIdx.x == 0 && blockIdx.x == 0) {
				prefix[blockIdx.x] = b_ptr_shared_reduction[0];       
				// Technically the aggregate is not needed
				agg[blockIdx.x] = b_ptr_shared_reduction[0];       
				__threadfence();
				flag[blockIdx.x] = 2;
			}
		}
		__syncthreads();

		int* exclusive_prefix_location = &b_ptr_shared_reduction[0];
		if (blockIdx.x > 0 && threadIdx.x == 0) {
			*exclusive_prefix_location = 0;
			int exclusive_prefix = 0;
			auto not_done = true;
			while (not_done) {
				*exclusive_prefix_location = 0;
				exclusive_prefix = 0;
				auto i = 1;
				auto t_flag = 0;
				auto t_agg = 0;
				auto t_prefix = 0;
				auto not_break_loop = true;
				while (i <= WINDOW && blockIdx.x - i >= 0 &&
							 not_break_loop) {
					// unsafe
					{
						t_flag = flag[blockIdx.x - i];
						__threadfence();
						t_agg = agg[blockIdx.x - i];
						t_prefix = prefix[blockIdx.x - i];
					}
					if (t_flag == 0) {
						not_break_loop = false;
					}
					if (t_flag == 1) {
						exclusive_prefix += t_agg;
					}
					if (t_flag == 2) {
						exclusive_prefix += t_prefix;
						not_break_loop = false;
						not_done = false;
					}
					i = i + 1;
				}
			}
			*exclusive_prefix_location = exclusive_prefix;
		}

    if (!threadIdx.x && blockIdx.x != 0) {
      // TODO move to function
      prefix[blockIdx.x] = b_ptr_shared_reduction[0] + agg[blockIdx.x];       
      __threadfence();
      flag[blockIdx.x] = 2;
    }
    __syncthreads();

		const auto foo = b_ptr_shared_input_copy;
		for (std::size_t d = THREADS_PER_BLOCK; d > 0; d = d / 2) {
			if (threadIdx.x < d) {
				(&(*foo))[(((threadIdx.x - 0) * ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d)) +
									 (((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d) - 1))] =
						(&(*foo))[(
								((threadIdx.x - 0) * ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d)) +
								(((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d) - 1))] +
						(&(*foo))[(
								((threadIdx.x - 0) * ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d)) +
								((THREADS_PER_BLOCK / d) - 1))];
			}
			__syncthreads();
		}
		
		if (threadIdx.x < 1) {
			foo[((threadIdx.x - 0) + ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) - 1))] = 0;
		}
		__syncthreads();

		for (std::size_t d = 1; d <= THREADS_PER_BLOCK; d = d * 2) {

			if (threadIdx.x < d) {
				const auto t = (&(
						*foo))[(((threadIdx.x - 0) * ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d)) +
										((THREADS_PER_BLOCK / d) - 1))];
				(&(*foo))[(((threadIdx.x - 0) * ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d)) +
									 ((THREADS_PER_BLOCK / d) - 1))] =
						(&(*foo))[(
								((threadIdx.x - 0) * ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d)) +
								(((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d) - 1))];
				(&(*foo))[(((threadIdx.x - 0) * ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d)) +
									 (((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d) - 1))] =
						(&(*foo))[(
								((threadIdx.x - 0) * ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d)) +
								(((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d) - 1))] +
						t;
			}
			__syncthreads();
		}

    // Parfor thread
    {
      if (blockIdx.x == 0) {
        for (int i = 0; i < ITEMS_PER_THREAD; i++) {
          t_ptr_input[i] = t_ptr_input[i] + t_ptr_shared_input[i];
        }
      } else {
        // TODO fix global colleasing
        for (int i = 0; i < ITEMS_PER_THREAD; i++) {
          t_ptr_input[i] = t_ptr_input[i] + t_ptr_shared_input[i];
        }
				for (int i = 0; i < ITEMS_PER_THREAD; i++) {
					t_ptr_input[i] = t_ptr_input[i] + b_ptr_shared_reduction[0];
				}
			}
    }
	}

}

Result exec() {
	int* c_input = (int*) malloc(SIZE_OF_INPUT);
	int* c_flag = (int*) malloc(SIZE_OF_STATUS_ARRS);	
	int* c_agg = (int*) malloc(SIZE_OF_STATUS_ARRS);	
	int* c_prefix = (int*) malloc(SIZE_OF_STATUS_ARRS);	


	int* g_input;
	int* g_flag;
	int* g_agg;
	int* g_prefix;

	for (int i = 0; i < AMOUNT_BLOCKS; i++) {
		c_flag[i] = 0;
		c_prefix[i] = 0;
		c_agg[i] = 0;
	}

	for (int i = 0; i < AMOUNT_ELEMS; i++) {
		c_input[i] = i % 11 + 1; //% 3;
	}

  int *gold = (int *)malloc(SIZE_OF_INPUT);
  scan_host(gold, c_input, AMOUNT_ELEMS);

	if (
		hipMalloc(&g_input, SIZE_OF_INPUT) |
		hipMalloc(&g_flag, SIZE_OF_STATUS_ARRS) |
		hipMalloc(&g_agg, SIZE_OF_STATUS_ARRS) |
		hipMalloc(&g_prefix, SIZE_OF_STATUS_ARRS) 
	) {
		printf("cannot alloc on gpu");
		exit(-1);
	}

	if (
		hipMemcpy(g_input, c_input, SIZE_OF_INPUT, hipMemcpyHostToDevice) |
		hipMemcpy(g_flag, c_flag, SIZE_OF_STATUS_ARRS, hipMemcpyHostToDevice) |
		hipMemcpy(g_prefix, c_prefix, SIZE_OF_STATUS_ARRS, hipMemcpyHostToDevice) |
		hipMemcpy(g_agg, c_agg, SIZE_OF_STATUS_ARRS, hipMemcpyHostToDevice)
	) {
		printf("cannot copy on gpu");
		exit(-1);
	}

		
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  scan_kernel<<<AMOUNT_BLOCKS, THREADS_PER_BLOCK, sizeof(int) * (THREADS_PER_BLOCK + ITEMS_PER_BLOCK) * 2>>>
		(g_input, g_flag, g_agg, g_prefix);
  hipEventRecord(stop);

  if (hipError_t error = hipDeviceSynchronize()) {
    std::cerr << "[!] Execution failed" << hipGetErrorName(error) << std::endl;
		exit(-1);
	}

	if (
		hipMemcpy(c_input, g_input, SIZE_OF_INPUT, hipMemcpyDeviceToHost) |
		hipMemcpy(c_flag, g_flag, SIZE_OF_STATUS_ARRS, hipMemcpyDeviceToHost) |
		hipMemcpy(c_prefix, g_prefix, SIZE_OF_STATUS_ARRS, hipMemcpyDeviceToHost) |
		hipMemcpy(c_agg, g_agg, SIZE_OF_STATUS_ARRS, hipMemcpyDeviceToHost)
	) {
    std::cerr << "[!] Memcpy failed" << std::endl;
		exit(-1);
	}

  if (arr_equal(gold, c_input, AMOUNT_ELEMS)) {
    std::cout << "[+] result is correct" << std::endl;
  } else {
    std::cerr << "[!] Result is not correct" << std::endl;
    for (int i = 0; i < AMOUNT_BLOCKS; i++) {
      printf("State %d got state %d and inclusive_prefix %d and agg %d\n", i,
             c_flag[i],
             c_prefix[i],
             c_agg[i]);
    }
		exit(-1);
	}

	hipFree(g_flag);
	hipFree(g_agg);
	hipFree(g_prefix);
	hipFree(g_input);

	free(c_flag);
	free(c_input);
	free(c_prefix);
	free(c_agg);
	free(gold);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  return Result{.time = milliseconds};
}


int main() {
  int iters = 250;

  Result results[iters];
  for (int i = 0; i < iters; i++) {
    results[i] = exec();
  }

  int sum = 0;
  FILE *fp;
  if((fp=fopen(CSV_OUTPUT_PATH, "w"))==NULL) {
    printf("cannot open.\n");
    exit(1);
  }

  for (int i = 0; i < iters; i++) {
    float time = results[i].time;
    sum += time;
    std::fprintf(fp, "%f,\n", time);
    // printf("time: %f\n", time);
  }
  std::fclose(fp);

  printf("success\n");
}
