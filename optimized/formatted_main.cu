#include "hip/hip_runtime.h"
#include "main.cuh"
#include "params.cuh"
#include <cstdio>
#include <iostream>

// CURSED ONE-File
void scan_host(int *dest, int *src, int n) {
  int *currDest = dest;
  int *currSrc = src;

  // Setting first value
  *currDest = *currSrc;

  // Moving the pointers through the array and using the last value to calc the
  // next value
  do {
    int nextVal = *(++currSrc) + *currDest;
    *(++currDest) = nextVal;
  } while (currDest != &dest[n]);
}

bool arr_equal(int *a, int *b, int n) {
  bool bad = false;
  for (int i = 0; i < n; i++) {
    if (abs(a[i] - b[i]) > 0.3) {
      std::cout << "I: " << i << " First val: " << a[i] << " Second Val" << b[i]
                << std::endl;
      return false;
      bad = true;
    }
  }
  return !bad;
}

__global__ void scan_kernel(int *const input, int *const flag, int *const agg,
                            int *const prefix) {
  __shared__ int b_shared_input_bkp[sizeof(int) * ITEMS_PER_BLOCK];
  __shared__ int s[sizeof(int) * (THREADS_PER_BLOCK + ITEMS_PER_BLOCK + 100)];

  // Parfor block
  {
    int *const b_ptr_input = &input[ITEMS_PER_BLOCK * blockIdx.x];
    int *const b_ptr_shared_reduction = &s[0];
    int *const b_ptr_shared_input_copy = &s[THREADS_PER_BLOCK];

    // for (int i = 0; i < ITEMS_PER_BLOCK; i += blockDim.x) {
    //   b_ptr_shared_input_copy[i + threadIdx.x] = b_ptr_input[i +
    //   threadIdx.x];
    // }
    for (int i = 0; i < ITEMS_PER_THREAD; i += 1) {
      b_ptr_shared_input_copy[i + (threadIdx.x * ITEMS_PER_THREAD)] =
          b_ptr_input[i + (threadIdx.x * ITEMS_PER_THREAD)];
      b_shared_input_bkp[i + (threadIdx.x * ITEMS_PER_THREAD)] =
          b_ptr_input[i + (threadIdx.x * ITEMS_PER_THREAD)];
    }
    __syncthreads();

    // Parfor thread in block
    int *const t_ptr_input = &b_ptr_input[threadIdx.x * ITEMS_PER_THREAD];
    int *const t_ptr_shared_reduction = &b_ptr_shared_reduction[threadIdx.x];
    int *const t_ptr_shared_input =
        &b_ptr_shared_input_copy[threadIdx.x * ITEMS_PER_THREAD];
    // PARFOR THREAD
    {
      // t_mem_cpy(t_ptr_shared_input, t_ptr_input);
      int sum = t_ptr_shared_input[0];
      for (int i = 1; i < ITEMS_PER_THREAD; i++) {
        sum = sum + t_ptr_shared_input[i];
      }
      *t_ptr_shared_reduction = sum;
    }
    __syncthreads();

    for (std::size_t k = (THREADS_PER_BLOCK / 2); k > 0; k = k / 2) {
      if (threadIdx.x < k) {
        b_ptr_shared_reduction[(threadIdx.x - 0)] =
            b_ptr_shared_reduction[(threadIdx.x - 0)] +
            b_ptr_shared_reduction[((threadIdx.x - 0) + k)];
      }
      __syncthreads();
    }

    if (threadIdx.x == 0) {
      if (blockIdx.x != 0) {
        agg[blockIdx.x] = b_ptr_shared_reduction[0];
        __threadfence();
        flag[blockIdx.x] = 1;
      } else if (threadIdx.x == 0 && blockIdx.x == 0) {
        prefix[blockIdx.x] = b_ptr_shared_reduction[0];
        // Technically the aggregate is not needed
        agg[blockIdx.x] = b_ptr_shared_reduction[0];
        __threadfence();
        flag[blockIdx.x] = 2;
      }
    }
    __syncthreads();

    int *const exclusive_prefix_location = &b_ptr_shared_reduction[0];
    const auto blockId = blockIdx.x;
    if (blockIdx.x > 0 && threadIdx.x == 0) {
      *exclusive_prefix_location = 0;
      int exclusive_prefix = 0;
      auto not_done = true;
      while (not_done) {
        *exclusive_prefix_location = 0;
        exclusive_prefix = 0;
        auto i = 1;
        auto t_flag = 0;
        auto t_agg = 0;
        auto t_prefix = 0;
        auto not_break_loop = true;
        while (i <= WINDOW && blockId - i >= 0 && not_break_loop) {
          // unsafe
          {
            t_flag = flag[blockId - i];
            __threadfence();
            t_agg = agg[blockId - i];
            t_prefix = prefix[blockId - i];
          }
          if (t_flag == 0) {
            not_break_loop = false;
          }
          if (t_flag == 1) {
            exclusive_prefix += t_agg;
          }
          if (t_flag == 2) {
            exclusive_prefix += t_prefix;
            not_break_loop = false;
            not_done = false;
          }
          i = i + 1;
        }
      }
      *exclusive_prefix_location = exclusive_prefix;
    }

    if (!threadIdx.x && blockIdx.x != 0) {
      // TODO move to function
      prefix[blockIdx.x] = b_ptr_shared_reduction[0] + agg[blockIdx.x];
      __threadfence();
      flag[blockIdx.x] = 2;
    }
    __syncthreads();

    const auto foo = b_ptr_shared_input_copy;
    for (std::size_t d = THREADS_PER_BLOCK; d > 0; d = d / 2) {
      if (threadIdx.x < d) {
        const auto baseThread = &foo[threadIdx.x * ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d)];
        //const auto t_ptr = &baseThread[(threadIdx.x * ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d))];
        baseThread[((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d) - 1] +=
            baseThread[((THREADS_PER_BLOCK / d) - 1)];
        baseThread[0] = 100;
      }
      __syncthreads();
    }

    if (threadIdx.x < 1) {
      foo[((threadIdx.x - 0) + ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) - 1))] =
          0;
    }
    __syncthreads();

    for (std::size_t d = 1; d <= THREADS_PER_BLOCK; d = d * 2) {

      if (threadIdx.x < d) {
        const auto t = (&(*foo))[(
            ((threadIdx.x - 0) * ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d)) +
            ((THREADS_PER_BLOCK / d) - 1))];
        (&(*foo))[(
            ((threadIdx.x - 0) * ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d)) +
            ((THREADS_PER_BLOCK / d) - 1))] =
            (&(*foo))[(((threadIdx.x - 0) *
                        ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d)) +
                       (((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d) - 1))];
        (&(*foo))[(
            ((threadIdx.x - 0) * ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d)) +
            (((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d) - 1))] =
            (&(*foo))[(((threadIdx.x - 0) *
                        ((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d)) +
                       (((THREADS_PER_BLOCK * ITEMS_PER_THREAD) / d) - 1))] +
            t;
      }
      __syncthreads();
    }

    // Parfor thread
    {
      if (blockIdx.x == 0) {
        for (int i = 0; i < ITEMS_PER_THREAD; i++) {
          t_ptr_input[i] = t_ptr_input[i] + t_ptr_shared_input[i];
        }
      } else {
        // TODO fix global colleasing
        for (int i = 0; i < ITEMS_PER_THREAD; i++) {
          t_ptr_input[i] =
              b_shared_input_bkp[(threadIdx.x * ITEMS_PER_THREAD) + i] +
              t_ptr_shared_input[i] + b_ptr_shared_reduction[0];
        }
      }
    }
  }
}

Result exec() {
  int *c_input = (int *)malloc(SIZE_OF_INPUT);
  int *c_flag = (int *)malloc(SIZE_OF_STATUS_ARRS);
  int *c_agg = (int *)malloc(SIZE_OF_STATUS_ARRS);
  int *c_prefix = (int *)malloc(SIZE_OF_STATUS_ARRS);

  int *g_input;
  int *g_flag;
  int *g_agg;
  int *g_prefix;

  for (int i = 0; i < AMOUNT_BLOCKS; i++) {
    c_flag[i] = 0;
    c_prefix[i] = 0;
    c_agg[i] = 0;
  }

  for (int i = 0; i < AMOUNT_ELEMS; i++) {
    c_input[i] = i % 11 + 1; //% 3;
  }

  int *gold = (int *)malloc(SIZE_OF_INPUT);
  scan_host(gold, c_input, AMOUNT_ELEMS);

  if (hipMalloc(&g_input, SIZE_OF_INPUT) |
      hipMalloc(&g_flag, SIZE_OF_STATUS_ARRS) |
      hipMalloc(&g_agg, SIZE_OF_STATUS_ARRS) |
      hipMalloc(&g_prefix, SIZE_OF_STATUS_ARRS)) {
    printf("cannot alloc on gpu");
    exit(-1);
  }

  if (hipMemcpy(g_input, c_input, SIZE_OF_INPUT, hipMemcpyHostToDevice) |
      hipMemcpy(g_flag, c_flag, SIZE_OF_STATUS_ARRS, hipMemcpyHostToDevice) |
      hipMemcpy(g_prefix, c_prefix, SIZE_OF_STATUS_ARRS,
                 hipMemcpyHostToDevice) |
      hipMemcpy(g_agg, c_agg, SIZE_OF_STATUS_ARRS, hipMemcpyHostToDevice)) {
    printf("cannot copy on gpu");
    exit(-1);
  }

  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  scan_kernel<<<AMOUNT_BLOCKS, THREADS_PER_BLOCK>>>(g_input, g_flag, g_agg,
                                                    g_prefix);
  hipEventRecord(stop);

  if (hipError_t error = hipDeviceSynchronize()) {
    std::cerr << "[!] Execution failed" << hipGetErrorName(error) << std::endl;
    exit(-1);
  }

  if (hipMemcpy(c_input, g_input, SIZE_OF_INPUT, hipMemcpyDeviceToHost) |
      hipMemcpy(c_flag, g_flag, SIZE_OF_STATUS_ARRS, hipMemcpyDeviceToHost) |
      hipMemcpy(c_prefix, g_prefix, SIZE_OF_STATUS_ARRS,
                 hipMemcpyDeviceToHost) |
      hipMemcpy(c_agg, g_agg, SIZE_OF_STATUS_ARRS, hipMemcpyDeviceToHost)) {
    std::cerr << "[!] Memcpy failed" << std::endl;
    exit(-1);
  }

  if (arr_equal(gold, c_input, AMOUNT_ELEMS)) {
    std::cout << "[+] result is correct" << std::endl;
  } else {
    std::cerr << "[!] Result is not correct" << std::endl;
    for (int i = 0; i < AMOUNT_BLOCKS; i++) {
      printf("State %d got state %d and inclusive_prefix %d and agg %d\n", i,
             c_flag[i], c_prefix[i], c_agg[i]);
    }
    exit(-1);
  }

  hipFree(g_flag);
  hipFree(g_agg);
  hipFree(g_prefix);
  hipFree(g_input);

  free(c_flag);
  free(c_input);
  free(c_prefix);
  free(c_agg);
  free(gold);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  return Result{.time = milliseconds};
}

int main() {
  int iters = 250;

  Result results[iters];
  for (int i = 0; i < iters; i++) {
    results[i] = exec();
  }

  int sum = 0;
  FILE *fp;
  if ((fp = fopen(CSV_OUTPUT_PATH, "w")) == NULL) {
    printf("cannot open.\n");
    exit(1);
  }

  for (int i = 0; i < iters; i++) {
    float time = results[i].time;
    sum += time;
    std::fprintf(fp, "%f,\n", time);
    // printf("time: %f\n", time);
  }
  std::fclose(fp);

  printf("success\n");
}
