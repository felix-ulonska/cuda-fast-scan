#include <iostream>

#include "../shared/shared.cuh"
#include "device.cuh"
#include "main.cuh"
#include "params.cuh"
#include <iostream>
#include <fstream>
#define VERSION "Baseline"

// I want to know if multiple compile units are the problem
#include "block.cu"
#include "device.cu"
#include "thread.cu"
#include "../shared/shared.cu"

Result exec() {
  int *input = (int*) malloc(SIZE_OF_INPUT);
  PartitionDescriptor *partition_descriptiors = (PartitionDescriptor*) malloc(SIZE_OF_PARTITION_DESCRIPTIORS);

  int *inputOnGpu;
  PartitionDescriptor *descOnGpu;

  // if (hipError_t error = hipMallocManaged(&input, SIZE_OF_INPUT)) {
  //   std::cerr << "[!] Cuda Malloc Managed for input failed with error"
  //             << hipGetErrorName(error) << std::endl;
  //   exit(EXIT_FAILURE);
  // }

  // if (hipError_t error = hipMallocManaged(&partition_descriptiors, SIZE_OF_PARTITION_DESCRIPTIORS)) {
  //   std::cerr << "[!] Cuda Malloc Managed for state failed with error"
  //             << hipGetErrorName(error) << std::endl;
  //   hipFree(input);
  //   exit(EXIT_FAILURE);
  // }

  init_array(input, AMOUNT_ELEMS);
  init_state_arr(partition_descriptiors, AMOUNT_BLOCKS);

  // I will assume that malloc will not fail.
  int *gold = (int *)malloc(SIZE_OF_INPUT);
  scan_host(gold, input, AMOUNT_ELEMS);

  // std::cout << "[+] Starting kernel..." << std::endl;

  if (hipMalloc(&inputOnGpu, SIZE_OF_INPUT) |
  hipMemcpy(inputOnGpu, input, SIZE_OF_INPUT, hipMemcpyHostToDevice) |
  hipMalloc(&descOnGpu, SIZE_OF_PARTITION_DESCRIPTIORS) | 
  hipMemcpy(descOnGpu, partition_descriptiors, SIZE_OF_PARTITION_DESCRIPTIORS, hipMemcpyHostToDevice)) {
    printf("BAD err");
    exit(1);
  }

  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  scan_kernel<<<AMOUNT_BLOCKS, THREADS_PER_BLOCK,
                  sizeof(int) * (THREADS_PER_BLOCK + ITEMS_PER_BLOCK) * 2>>>(inputOnGpu,
                                                        descOnGpu);
  hipEventRecord(stop);

  if (hipError_t error = hipDeviceSynchronize()) {
    std::cerr << "[!] Execution failed" << hipGetErrorName(error) << std::endl;
    hipFree(input);
    hipFree(partition_descriptiors);
    free(gold);
    exit(EXIT_FAILURE);
  }

  hipMemcpy(input, inputOnGpu, SIZE_OF_INPUT, hipMemcpyDeviceToHost);
  hipMemcpy(partition_descriptiors, descOnGpu, SIZE_OF_PARTITION_DESCRIPTIORS, hipMemcpyDeviceToHost);

  if (hipError_t error = hipEventSynchronize(stop)) {
    std::cerr << "[!] Event Sync failed" << hipGetErrorName(error)
              << std::endl;
    hipFree(input);
    hipFree(partition_descriptiors);
    free(gold);
    exit(EXIT_FAILURE);
  };

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // printf("took %f ms\n", milliseconds);

  if (arr_equal(gold, input, AMOUNT_ELEMS)) {
    // std::cout << "[+] result is correct" << std::endl;
  } else {
    std::cerr << "[!] Result is not correct" << std::endl;

    for (int i = 0; i < AMOUNT_BLOCKS; i++) {
      printf("State %d got state %d and inclusive_prefix %d and agg %d\n", i,
             partition_descriptiors[i].flag,
             partition_descriptiors[i].inclusive_prefix,
             partition_descriptiors[i].aggregate);
    }

    hipFree(input);
    hipFree(partition_descriptiors);
    free(gold);
    exit(EXIT_FAILURE);
  }
  free(input);
  free(partition_descriptiors);
  hipFree(inputOnGpu);
  hipFree(descOnGpu);
  free(gold);
  return Result{.time = milliseconds};
}

int main() {
  int iters = 250;

  Result results[iters];
  for (int i = 0; i < iters; i++) {
    results[i] = exec();
  }

  int sum = 0;
  FILE *fp;
  if((fp=fopen(CSV_OUTPUT_PATH, "w"))==NULL) {
    printf("cannot open.\n");
    exit(1);
  }

  for (int i = 0; i < iters; i++) {
    float time = results[i].time;
    sum += time;
    std::fprintf(fp, "%f,\n", time);
    // printf("time: %f\n", time);
  }
  std::fclose(fp);

  printf("success\n");
}
