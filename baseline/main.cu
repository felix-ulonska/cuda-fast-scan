#include <iostream>

#include "../shared/shared.cuh"
#include "device.cuh"
#include "main.cuh"

#define VERSION "Baseline"

Result exec() {
  int *input;
  PartitionDescriptor *partition_descriptiors;

  if (hipError_t error = hipMallocManaged(&input, SIZE_OF_INPUT)) {
    std::cerr << "[!] Cuda Malloc Managed for input failed with error"
              << hipGetErrorName(error) << std::endl;
    exit(EXIT_FAILURE);
  }

  if (hipError_t error = hipMallocManaged(&partition_descriptiors, SIZE_OF_PARTITION_DESCRIPTIORS)) {
    std::cerr << "[!] Cuda Malloc Managed for state failed with error"
              << hipGetErrorName(error) << std::endl;
    hipFree(input);
    exit(EXIT_FAILURE);
  }

  init_array(input, AMOUNT_ELEMS);
  init_state_arr(partition_descriptiors, AMOUNT_BLOCKS);

  // I will assume that malloc will not fail.
  int *gold = (int *)malloc(SIZE_OF_INPUT);
  scan_host(gold, input, AMOUNT_ELEMS);

  // std::cout << "[+] Starting kernel..." << std::endl;

  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  scan_kernel<<<AMOUNT_BLOCKS, THREADS_PER_BLOCK,
                  sizeof(int) * (THREADS_PER_BLOCK + ITEMS_PER_BLOCK)>>>(input,
                                                        partition_descriptiors);
  hipEventRecord(stop);

  if (hipError_t error = hipDeviceSynchronize()) {
    std::cerr << "[!] Execution failed" << hipGetErrorName(error) << std::endl;
    hipFree(input);
    hipFree(partition_descriptiors);
    free(gold);
    exit(EXIT_FAILURE);
  }

  if (hipError_t error = hipEventSynchronize(stop)) {
    std::cerr << "[!] Event Sync failed" << hipGetErrorName(error)
              << std::endl;
    hipFree(input);
    hipFree(partition_descriptiors);
    free(gold);
    exit(EXIT_FAILURE);
  };

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // printf("took %f ms\n", milliseconds);

  if (arr_equal(gold, input, AMOUNT_ELEMS)) {
    // std::cout << "[+] result is correct" << std::endl;
  } else {
    std::cerr << "[!] Result is not correct" << std::endl;

    for (int i = 0; i < AMOUNT_BLOCKS; i++) {
      printf("State %d got state %d and inclusive_prefix %d and agg %d\n", i,
             partition_descriptiors[i].flag,
             partition_descriptiors[i].inclusive_prefix,
             partition_descriptiors[i].aggregate);
    }

    hipFree(input);
    hipFree(partition_descriptiors);
    free(gold);
    exit(EXIT_FAILURE);
  }
  hipFree(input);
  hipFree(partition_descriptiors);
  free(gold);
  return Result{.time = milliseconds};
}

int main() {
  int iters = 1;

  Result results[iters];
  for (int i = 0; i < iters; i++) {
    results[i] = exec();
    printf(".");
  }
  printf("\n");

  int sum = 0;
  for (int i = 0; i < iters; i++) {
    int time = results[i].time;
    sum += time;

    // printf("time: %f\n", time);
  }

  printf("Avg: %d", sum / iters);
}
